#include "hip/hip_runtime.h"
#include "Kernels.cuh"
#include "../Simulation.h"

using namespace SPH;

//////////////////////////////////////////////////////////////////
// Helper host methods 
//////////////////////////////////////////////////////////////////

KernelData::KernelData()
{
	CudaHelper::CudaMalloc(&d_W, PRECOMPUTED_KERNEL_SIZE);
	CudaHelper::CudaMalloc(&d_gradW, PRECOMPUTED_KERNEL_SIZE + 1);
}

KernelData::~KernelData()
{
	CudaHelper::CudaFree(d_W);
	CudaHelper::CudaFree(d_gradW);
}

void updateKernelData(KernelData &data)
{
	data.radius = PrecomputedKernel<CubicKernel, PRECOMPUTED_KERNEL_SIZE>::getRadius();
	data.invStepSize = PrecomputedKernel<CubicKernel, PRECOMPUTED_KERNEL_SIZE>::getInvStepSize();
	data.radius2 = data.radius * data.radius;

	CudaHelper::MemcpyHostToDevice(PrecomputedKernel<CubicKernel, PRECOMPUTED_KERNEL_SIZE>::getWeightField(), data.d_W, PRECOMPUTED_KERNEL_SIZE);
	CudaHelper::MemcpyHostToDevice(PrecomputedKernel<CubicKernel, PRECOMPUTED_KERNEL_SIZE>::getGradField(), data.d_gradW, PRECOMPUTED_KERNEL_SIZE + 1);
}

//////////////////////////////////////////////////////////////////
//Kernels for all methods 
//////////////////////////////////////////////////////////////////

__device__
Real kernelWeightPrecomputed(const Vector3r &r, const KernelData* const data)
{
	Real res = 0.0;
	const Real r2 = r.squaredNorm();
	if (r2 <= data->radius2)
	{
		const Real rl = sqrt(r2);
		//const unsigned int pos = std::min<unsigned int>((unsigned int)(rl * data->invStepSize), PRECOMPUTED_KERNEL_SIZE-2u);
		unsigned int pos = 0;
		if(static_cast<unsigned int>(rl * data->invStepSize) < PRECOMPUTED_KERNEL_SIZE-2u)
			pos = static_cast<unsigned int>(rl * data->invStepSize);
		else
			pos = PRECOMPUTED_KERNEL_SIZE-2u;
		res = static_cast<Real>(0.5)*(data->d_W[pos] + data->d_W[pos+1]);
	}
	return res;
}

__device__
Vector3r gradKernelWeightPrecomputed(const Vector3r &r, const KernelData* const data)
{
	Vector3r res;
	const Real rl = r.norm(); // rl / radius = > 0 - 1, texturSpeicher
	if (rl <= data->radius)
	{
		//const Real rl = sqrt(r2);
		//const unsigned int pos = static_cast<unsigned int>(fminf(static_cast<unsigned int>(rl * data->invStepSize), PRECOMPUTED_KERNEL_SIZE-1u));
		unsigned int pos = 0;
		if(static_cast<unsigned int>(rl * data->invStepSize) < PRECOMPUTED_KERNEL_SIZE-1u)
			pos = static_cast<unsigned int>(rl * data->invStepSize);
		else
			pos = PRECOMPUTED_KERNEL_SIZE-1u;
		res = 0.5*(data->d_gradW[pos] + data->d_gradW[pos + 1]) * r; // ersetzbar
	}
	else
		res.setZero();

	return res;
}

__device__
Real kernelWeight(const Vector3r& rin, const Real m_radius)
{
	const Real r = sqrt(rin[0] * rin[0] + rin[1] * rin[1] + rin[2] * rin[2]);
	const Real pi = 3.14159265358979323846;

	const Real h3 = m_radius*m_radius*m_radius;
	Real m_k = static_cast<Real>(8.0) / (pi*h3);
	Real m_l = static_cast<Real>(48.0) / (pi*h3);

	Real res = 0.0;
	const Real q = r / m_radius;

	if (q <= 1.0)
	{
		if (q <= 0.5)
		{
			const Real q2 = q*q;
			const Real q3 = q2*q;
			res = m_k * (static_cast<Real>(6.0)*q3 - static_cast<Real>(6.0)*q2 + static_cast<Real>(1.0));
		}
		else
		{
			res = m_k * (static_cast<Real>(2.0)*pow(static_cast<Real>(1.0) - q, 3));
		}
	}
	return res;
}

__device__
Vector3r gradKernelWeight(const Vector3r &rin, const Real m_radius)
{

	const Real pi = 3.14159265358979323846;
	const Real h3 = m_radius*m_radius*m_radius;
	const Real m_l = static_cast<Real>(48.0) / (pi*h3);

	Vector3r res;
	const Real rl = sqrt(rin[0] * rin[0] + rin[1] * rin[1] + rin[2] * rin[2]);
	const Real q = rl / m_radius;
	if ((rl > 1.0e-6) && (q <= 1.0))
	{
		const Vector3r gradq = rin * (static_cast<Real>(1.0) / (rl*m_radius));
		if (q <= 0.5)
		{
			res = m_l*q*((Real) 3.0*q - static_cast<Real>(2.0))*gradq;
		}
		else
		{
			const Real factor = static_cast<Real>(1.0) - q;
			res = m_l*(-factor*factor)*gradq;
		}
	}
	else
		res.setZero();

	return res;
}


__device__
void addForce(const Vector3r &pos, const Vector3r &f, /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, 
	const Vector3r* const rigidBodyPositions, const uint* const forcesPerThreadIndices, const uint* const torquesPerThreadIndices, const uint index, const int id)
{
	#ifdef _OPENMP
	int tid = id;
	#else
	int tid = 0;
	#endif
	forcesPerThread[forcesPerThreadIndices[index] + tid] += f;
	torquesPerThread[torquesPerThreadIndices[index] + tid] += (pos - rigidBodyPositions[index]).cross(f);
}


__global__
void computeDensitiesGPU(/*out*/ Real* const densities, const Real* const volumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, 
	const uint* const fmIndices, const Real* const densities0, const Real W_zero, const KernelData* const kernelData, 
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
 	// Boundary: Akinci2012
	const uint i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= numParticles)
		return;

	extern __shared__ Real densities_tmp[];

	Real &density = densities_tmp[threadIdx.x];

	density = volumes[fluidModelIndex] * W_zero;
	const double3 &xi = particles[fluidModelIndex][i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		density += volumes[pid] * kernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
	)
	

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
  forall_boundary_neighborsGPU(
		density += boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] *  kernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
	)

	density *= densities0[fluidModelIndex];

	densities[fmIndices[fluidModelIndex] + i] = densities_tmp[threadIdx.x];
} 


//////////////////////////////////////////////////////////////////
//Kernels for the WCPSH method 
//////////////////////////////////////////////////////////////////

__global__
void clearAccelerationsGPU(Real* masses, Vector3r* accelerations, const Vector3r grav, const uint numActiveParticles)
{
 	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= numActiveParticles)
		return;

	// Clear accelerations of dynamic particles
	if (masses[i] != 0.0)
	{
		Vector3r &a = accelerations[i];
		a = grav;
	}
}

__global__
void updatePressureGPU(Real* const densities, const uint* const fmIndices, Real* const pressures, const Real* const densities0, const Real m_stiffness, const Real m_exponent,
	const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= numParticles)
		return;
	
	Real &density = densities[fmIndices[fluidModelIndex] + i];
	density = max(density, densities0[fluidModelIndex]);
	pressures[fmIndices[fluidModelIndex] + i] = m_stiffness * (pow(density / densities0[fluidModelIndex], m_exponent) - static_cast<Real>(1.0));
}

__global__
void computePressureAccelsGPU( /* output */ Vector3r* const pressureAccels, /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, const uint* const forcesPerThreadIndices, 
	const uint* const torquesPerThreadIndices, const Real* const densities, const Real* const densities0, const uint* const fmIndices, const Real* const pressures, const Real* const masses, 
	const Vector3r* const rigidBodyPositions, const Real* const volumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, const bool* const isDynamic, const int tid, const KernelData* kernelData,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
   const uint i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= numParticles)
		return;

	extern __shared__ Vector3r pressureAccels_tmp[];

	const double3 &xi = particles[fluidModelIndex][i];

	const Real density_i = densities[fmIndices[fluidModelIndex] + i];

	pressureAccels_tmp[threadIdx.x] = Vector3r(0, 0, 0);
	Vector3r &ai = pressureAccels_tmp[threadIdx.x];

	const Real dpi = pressures[fmIndices[fluidModelIndex] + i] / (density_i*density_i);
	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Real density_j = densities[fmIndices[pid] + neighborIndex] * densities0[fluidModelIndex] / densities0[pid];
		const Real dpj = pressures[fmIndices[pid] + neighborIndex] / (density_j*density_j);
		ai -= densities0[fluidModelIndex] * volumes[pid] * (dpi + dpj) * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	const Real dpj = pressures[fmIndices[fluidModelIndex] + i] / (densities0[fluidModelIndex] * densities0[fluidModelIndex]);
	forall_boundary_neighborsGPU(
		const Vector3r a = densities0[fluidModelIndex] * boundaryVolumes[fmIndices[pid - nFluids] + neighborIndex] * (dpi + dpj) * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
		ai -= a;
		if(isDynamic[pid - nFluids])
		{
			addForce(Vector3r(xj.x, xj.y, xj.z), masses[i] * a, forcesPerThread, torquesPerThread, rigidBodyPositions, forcesPerThreadIndices, torquesPerThreadIndices, pid - nFluids, tid);
		}
	)

	pressureAccels[i] = pressureAccels_tmp[threadIdx.x];
}

__global__ 
void updatePosPressureAccelPressureAccel(Vector3r* const positions, Vector3r* const velocities, Vector3r* const accelerations,
	const Vector3r* const pressureAccels, const Real h, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= numParticles)
		return;
	
	accelerations[i] += pressureAccels[i];
	velocities[i] += accelerations[i] * h;
	positions[i] += velocities[i] * h;
	
}


//////////////////////////////////////////////////////////////////
//Kernels for the DFSPH method 
//////////////////////////////////////////////////////////////////

__global__ 
void computeDFSPHFactors(/* out */ Real* factors, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, const KernelData* const kernelData, 
	const unsigned int* fmIndices, const Real* fmVolumes, const Real eps,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;
	
	Real &factor = factors[fmIndices[fluidModelIndex] + i];
	factor = 0.0;

	//////////////////////////////////////////////////////////////////////////
	// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
	//////////////////////////////////////////////////////////////////////////

	const double3 xi = particles[fluidModelIndex][i];
	Real sum_grad_p_k = 0.0;
	Vector3r grad_p_i;
	grad_p_i.setZero();

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
forall_fluid_neighborsGPU(
	const Vector3r grad_p_j = -fmVolumes[fluidModelIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
	sum_grad_p_k += grad_p_j.squaredNorm();
	grad_p_i -= grad_p_j;
)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	forall_boundary_neighborsGPU(
		const Vector3r grad_p_j = -boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
		grad_p_i -= grad_p_j;
	)

	sum_grad_p_k += grad_p_i.squaredNorm();

	//////////////////////////////////////////////////////////////////////////
	// Compute pressure stiffness denominator
	//////////////////////////////////////////////////////////////////////////
	if (sum_grad_p_k > eps)
		factor = -static_cast<Real>(1.0) / (sum_grad_p_k);
	else
		factor = 0.0;
}


 __global__
void computeDensityChanges(/*out*/ Real* const densitiesAdv, const Vector3r* const fmVelocities, const Vector3r* const bmVelocities, const uint* const fmIndices, 
	const Real* const fmVolumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, const KernelData* const kernelData,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	Real &densityAdv = densitiesAdv[fmIndices[fluidModelIndex] + i];	
	const double3 &xi = particles[fluidModelIndex][i];
	const Vector3r &vi = fmVelocities[fmIndices[fluidModelIndex] + i];

	densityAdv = 0.0;
	unsigned int numNeighbors = 0;

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Vector3r &vj = fmVelocities[fmIndices[pid] + neighborIndex];
		densityAdv += fmVolumes[pid] * (vi - vj).dot(gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData));
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	forall_boundary_neighborsGPU(
		const Vector3r &vj = bmVelocities[boundaryVolumeIndices[pid - nFluids] + neighborIndex];
		densityAdv += boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * (vi - vj).dot(gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData));
	)
	
	// only correct positive divergence
	densityAdv = max(densityAdv, static_cast<Real>(0.0));

	for (unsigned int pid = 0; pid < nPointSets; pid++)
	{
		const uint neighborsetIndex = neighborPointsetIndices[fluidModelIndex] + pid;
		numNeighbors += neighborCounts[neighborsetIndex][i];
	}

	// in case of particle deficiency do not perform a divergence solve
	if (numNeighbors < 20)
		densityAdv = 0.0;
}

__global__
void computeDensityAdvs(/*out*/ Real* const densitiesAdv, const Real* const fmDensities, const Vector3r* const fmVelocities, const Vector3r* const bmVelocities, const uint* const fmIndices, 
	const Real* const fmVolumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, const Real* const densities0, const Real h, const KernelData* const kernelData,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	Real &densityAdv = densitiesAdv[fmIndices[fluidModelIndex] + i];
	const Real &density = fmDensities[fmIndices[fluidModelIndex] + i];
	const double3 &xi = particles[fluidModelIndex][i];
	const Vector3r &vi = fmVelocities[fmIndices[fluidModelIndex] + i];
	Real delta = 0.0;

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Vector3r &vj = fmVelocities[fmIndices[pid] + neighborIndex];
		delta += fmVolumes[pid] * (vi - vj).dot(gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData));
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	forall_boundary_neighborsGPU(
		const Vector3r &vj = bmVelocities[boundaryVolumeIndices[pid - nFluids] + neighborIndex];
		delta += boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * (vi - vj).dot(gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData));
	)
	
	densityAdv = density / densities0[fluidModelIndex] + h*delta;
	densityAdv = max(densityAdv, static_cast<Real>(1.0));
}

__global__
void warmstartDivergenceSolveKappaV(/*out*/ Real* const kappaV, const uint* const fmIndices, const Real* const densities0, const Real invH, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;
	
	kappaV[fmIndices[fluidModelIndex] + i] = static_cast<Real>(0.5) * max( kappaV[fmIndices[fluidModelIndex] + i] * invH, -static_cast<Real>(0.5) * densities0[fluidModelIndex] * densities0[fluidModelIndex]);
}

__global__
void divergenceSolveWarmstart( /*out*/ Vector3r* const fmVelocities, /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, 
	const uint* const forcesPerThreadIndices, const uint* const torquesPerThreadIndices, const Vector3r* const rigidBodyPositions, const Real* const kappaV,
	const uint* const fmIndices, const Real* const masses, const Real* const fmVolumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, 
	const Real* const densities0, const bool* const isDynamic, const int tid, const Real h, const KernelData* const kernelData, const Real eps,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles || numParticles == 0)
		return;

	const Real invH = static_cast<Real>(1.0) / h;

	Vector3r &vel = fmVelocities[fmIndices[fluidModelIndex] + i];
	const Real ki = kappaV[fmIndices[fluidModelIndex] + i];
	const double3 &xi = particles[fluidModelIndex][i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Real kj = kappaV[fmIndices[pid] + neighborIndex];

		const Real kSum = (ki + densities0[pid] / densities0[fluidModelIndex] * kj);
		if (fabsf(kSum) > eps)
		{
			const Vector3r grad_p_j = -fmVolumes[pid] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			vel -= h * kSum * grad_p_j;					// ki, kj already contain inverse density
		}
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if (fabsf(ki) > eps)
	{
		forall_boundary_neighborsGPU(
			const Vector3r grad_p_j = -boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			const Vector3r velChange = -h * (Real) 1.0 * ki * grad_p_j;				// kj already contains inverse density
			vel += velChange;
			addForce(Vector3r(xj.x, xj.y, xj.z), -masses[fmIndices[fluidModelIndex] + i] * velChange * invH, forcesPerThread, torquesPerThread, rigidBodyPositions, forcesPerThreadIndices, torquesPerThreadIndices, pid - nFluids, tid);
		)
	}
}


__global__
void multiplyRealWithConstant(/*out*/ Real* const input, const uint* const fmIndices, const Real f, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	input[fmIndices[fluidModelIndex] + i] *= f;
}

__global__
void setRealToZero(/*out*/ Real* const input, const uint* const fmIndices, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	input[fmIndices[fluidModelIndex] + i] = 0.0;
}

__global__
void divergenceSolveUpdateKappaV(/*out*/ Real* const kappaV, const Real* const densitiesAdv, const Real* const factors, const uint* const fmIndices, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	const Real b_i = densitiesAdv[fmIndices[fluidModelIndex] + i];
	const Real ki = b_i * factors[fmIndices[fluidModelIndex] + i];
	kappaV[fmIndices[fluidModelIndex] + i] += ki;
}

__global__
void pressureSolveUpdateKappa(/*out*/ Real* const kappa, const Real* const densitiesAdv, const Real* const factors, const uint* const fmIndices, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	const Real b_i = densitiesAdv[fmIndices[fluidModelIndex] + i]  - static_cast<Real>(1.0);
	const Real ki = b_i * factors[fmIndices[fluidModelIndex] + i];
	kappa[fmIndices[fluidModelIndex] + i] += ki;
}

__global__ 
void updateFluidVelocities( /*out*/ Vector3r* const fmVelocities, /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, 
	const uint* const forcesPerThreadIndices, const uint* const torquesPerThreadIndices, const Vector3r* const rigidBodyPositions, const Real* const densitiesAdv, const Real* const factors, 
	const uint* const fmIndices, const Real* const masses, const Real* const fmVolumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, 
	const Real* const densities0, const bool* const isDynamic, const int tid, const Real h, const Real invH, const KernelData* const kernelData, const Real eps,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
 	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	const Real b_i = densitiesAdv[fmIndices[fluidModelIndex] + i];
	const Real ki = b_i * factors[fmIndices[fluidModelIndex] + i];

	Vector3r &v_i = fmVelocities[fmIndices[fluidModelIndex] + i];
	const double3 &xi = particles[fluidModelIndex][i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Real b_j = densitiesAdv[fmIndices[pid] + neighborIndex];
		const Real kj = b_j * factors[fmIndices[pid] + neighborIndex];

		const Real kSum = ki + densities0[pid] / densities0[fluidModelIndex] * kj;
		if(fabsf(kSum) > eps)
		{
			const Vector3r grad_p_j = -fmVolumes[pid] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			v_i -= h * kSum * grad_p_j; // ki, kj already contain inverse density
		}
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if(fabsf(ki) > eps)
	{
		forall_boundary_neighborsGPU(
			const Vector3r grad_p_j = -boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			const Vector3r velChange = -h * (Real) 1.0 * ki * grad_p_j;	// kj already contains inverse density
			v_i += velChange;
			addForce(Vector3r(xj.x, xj.y, xj.z), -masses[fmIndices[fluidModelIndex] + i] * velChange * invH, forcesPerThread, torquesPerThread, rigidBodyPositions, forcesPerThreadIndices, torquesPerThreadIndices, pid - nFluids, tid);
		)
	}
} 

__global__ 
void pressureSolveUpdateFluidVelocities( /*out*/ Vector3r* const fmVelocities, /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, 
	const uint* const forcesPerThreadIndices, const uint* const torquesPerThreadIndices, const Vector3r* const rigidBodyPositions, const Real* const densitiesAdv, const Real* const factors, 
	const uint* const fmIndices, const Real* const masses, const Real* const fmVolumes, const Real* const boundaryVolumes, const uint* const boundaryVolumeIndices, 
	const Real* const densities0, const bool* const isDynamic, const int tid, const Real h, const Real invH, const KernelData* const kernelData, const Real eps,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
 	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	const Real b_i = densitiesAdv[fmIndices[fluidModelIndex] + i] - static_cast<Real>(1.0);
	const Real ki = b_i * factors[fmIndices[fluidModelIndex] + i];

	Vector3r &v_i = fmVelocities[fmIndices[fluidModelIndex] + i];
	const double3 &xi = particles[fluidModelIndex][i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	forall_fluid_neighborsGPU(
		const Real b_j = densitiesAdv[fmIndices[pid] + neighborIndex] - static_cast<Real>(1.0);
		const Real kj = b_j * factors[fmIndices[pid] + neighborIndex];

		const Real kSum = ki + densities0[pid] / densities0[fluidModelIndex] * kj;
		if(fabsf(kSum) > eps)
		{
			const Vector3r grad_p_j = -fmVolumes[pid] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			v_i -= h * kSum * grad_p_j; // ki, kj already contain inverse density
		}
	)

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	if(fabsf(ki) > eps)
	{
		forall_boundary_neighborsGPU(
			const Vector3r grad_p_j = -boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
			const Vector3r velChange = -h * (Real) 1.0 * ki * grad_p_j;	// kj already contains inverse density
			v_i += velChange;
			addForce(Vector3r(xj.x, xj.y, xj.z), -masses[fmIndices[fluidModelIndex] + i] * velChange * invH, forcesPerThread, torquesPerThread, rigidBodyPositions, forcesPerThreadIndices, torquesPerThreadIndices, pid - nFluids, tid);
		)
	}
} 

__global__
void updateDensityErrorDivergence(/*out*/ Real* const density_errors, const Real* const densitiesAdv, const Real* const densities0, const uint* const fmIndices,
	const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	//density_errors[fluidModelIndex] += densities0[fluidModelIndex] * densitiesAdv[fmIndices[fluidModelIndex] + i];
	density_errors[0] += densities0[fluidModelIndex] * densitiesAdv[fmIndices[fluidModelIndex] + i];
}

__global__
void warmstartPressureSolveKappa(/*out*/ Real* kappa, const uint* const fmIndices, const Real* const densities0, const Real invH2, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;
	
	kappa[fmIndices[fluidModelIndex] + i] = max( kappa[fmIndices[fluidModelIndex] + i] * invH2, -static_cast<Real>(0.5) * densities0[fluidModelIndex] * densities0[fluidModelIndex]);
}

__global__
void pressureSolveWarmstart(/*out*/ Vector3r* const fmVelocities , /* output */ Vector3r* const forcesPerThread, /* output */ Vector3r* const torquesPerThread, 
	const uint* const forcesPerThreadIndices, const uint* const torquesPerThreadIndices, const Vector3r* const rigidBodyPositions,const Real* const kappa, 
	const Real* const densitiesAdv, const Real* const masses, const Real* const fmVolumes, const uint* const fmIndices, const Real* const boundaryVolumes, 
	const uint* const boundaryVolumeIndices, const Real* const densities0, const bool* const isDynamic, const int tid, const Real h, const Real eps, const KernelData* const kernelData,
	/*start of forall-parameters*/ double3** particles, uint** neighbors, uint** neighborCounts, uint** neighborOffsets, 
  uint* neighborPointsetIndices, const uint nFluids, const uint nPointSets, const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	if(densitiesAdv[fmIndices[fluidModelIndex] + i] > densities0[fluidModelIndex])
	{
		const Real invH = static_cast<Real>(1.0) / h;

		Vector3r &vel = fmVelocities[fmIndices[fluidModelIndex] + i];
		const Real &ki = kappa[fmIndices[fluidModelIndex] + i];
		const double3 &xi = particles[fluidModelIndex][i];

		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		forall_fluid_neighborsGPU(
			const Real kj = kappa[fmIndices[pid] + neighborIndex];

			const Real kSum = (ki + densities0[pid] / densities0[fluidModelIndex] * kj);
			if (fabsf(kSum) > eps)
			{
				const Vector3r grad_p_j = -fmVolumes[pid] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
				vel -= h * kSum * grad_p_j;					// ki, kj already contain inverse density
			}
		)

		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
		if (fabsf(ki) > eps)
		{
			forall_boundary_neighborsGPU(
				const Vector3r grad_p_j = -boundaryVolumes[boundaryVolumeIndices[pid - nFluids] + neighborIndex] * gradKernelWeightPrecomputed(Vector3r(xi.x - xj.x, xi.y - xj.y, xi.z - xj.z), kernelData);
				const Vector3r velChange = -h * (Real) 1.0 * ki * grad_p_j;				// kj already contains inverse density
				vel += velChange;
				addForce(Vector3r(xj.x, xj.y, xj.z), -masses[fmIndices[fluidModelIndex] + i] * velChange * invH, forcesPerThread, torquesPerThread, rigidBodyPositions, forcesPerThreadIndices, torquesPerThreadIndices, pid - nFluids, tid);
			)
		}
	}
}

__global__
void updateDensityErrorPressureSolve(/*out*/ Real* const density_error, const Real* const densitiesAdv, const Real* const densities0, const uint* const fmIndices,
	const uint fluidModelIndex, const uint numParticles)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= numParticles)
		return;

	//density_errors[fluidModelIndex] += densities0[fluidModelIndex] * densitiesAdv[fmIndices[fluidModelIndex] + i];
	density_error[0] += densities0[fluidModelIndex] * densitiesAdv[fmIndices[fluidModelIndex] + i] - densities0[fluidModelIndex];
}